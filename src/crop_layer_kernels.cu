#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

#include "crop_layer.h"
#include "utils.h"
#include "dark_cuda.h"
#include "image.h"

__device__ float get_pixel_kernel(float *image, int w, int h, int x, int y, int c)
{
    if(x < 0 || x >= w || y < 0 || y >= h) return 0;
    return image[x + w*(y + c*h)];
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    auto r = rgb.x;
    auto g = rgb.y;
    auto b = rgb.z;

    float h, s;
    const auto max = (r > g) ? ( (r > b) ? r : b) : ( (g > b) ? g : b);
    const auto min = (r < g) ? ( (r < b) ? r : b) : ( (g < b) ? g : b);
    const auto delta = max - min;
    float v = max;
    if(max == 0){
        s = 0;
        h = -1;
    }else{
        s = delta/max;
        if(r == max){
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        if (h < 0) h += 6;
    }
    return make_float3(h, s, v);
}

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    const auto h = hsv.x;
    const auto s = hsv.y;
    const auto v = hsv.z;

    float r, g, b;

    if (s == 0) {
        r = g = b = v;
    } else {
        const auto index = (int) floorf(h);
        float f = h - index;
        float p = v * (1 - s);
        float q = v * (1 - s * f);
        float t = v * (1 - s * (1 - f));
        if(index == 0){
            r = v; g = t; b = p;
        } else if(index == 1){
            r = q; g = v; b = p;
        } else if(index == 2){
            r = p; g = v; b = t;
        } else if(index == 3){
            r = p; g = q; b = v;
        } else if(index == 4){
            r = t; g = p; b = v;
        } else {
            r = v; g = p; b = q;
        }
    }
    r = (r < 0) ? 0 : ((r > 1) ? 1 : r);
    g = (g < 0) ? 0 : ((g > 1) ? 1 : g);
    b = (b < 0) ? 0 : ((b > 1) ? 1 : b);
    return make_float3(r, g, b);
}

__device__ float bilinear_interpolate_kernel(float *image, int w, int h, float x, float y, int c)
{
    const auto ix = (int) floorf(x);
    const auto iy = (int) floorf(y);

    const auto dx = x - ix;
    const auto dy = y - iy;

    const auto val = (1-dy) * (1-dx) * get_pixel_kernel(image, w, h, ix, iy, c) +
        dy     * (1-dx) * get_pixel_kernel(image, w, h, ix, iy+1, c) +
        (1-dy) *   dx   * get_pixel_kernel(image, w, h, ix+1, iy, c) +
        dy     *   dx   * get_pixel_kernel(image, w, h, ix+1, iy+1, c);
    return val;
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    auto size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;
    id /= h;
    const auto rshift = rand[0];
    const auto gshift = rand[1];
    const auto bshift = rand[2];
    float r0 = rand[8*id + 0];
    float r1 = rand[8*id + 1];
    float r2 = rand[8*id + 2];
    float r3 = rand[8*id + 3];

    saturation = r0*(saturation - 1) + 1;
    saturation = (r1 > .5) ? 1./saturation : saturation;
    exposure = r2*(exposure - 1) + 1;
    exposure = (r3 > .5) ? 1./exposure : exposure;

    size_t offset = id * h * w * 3;
    image += offset;
    const float r = image[x + w*(y + h*0)];
    const float g = image[x + w*(y + h*1)];
    const float b = image[x + w*(y + h*2)];
    float3 rgb = make_float3(r,g,b);
    if(train){
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }
    image[x + w*(y + h*0)] = rgb.x*scale + translate + (rshift - .5)*shift;
    image[x + w*(y + h*1)] = rgb.y*scale + translate + (gshift - .5)*shift;
    image[x + w*(y + h*2)] = rgb.z*scale + translate + (bshift - .5)*shift;
}

__global__ void forward_crop_layer_kernel(float *input, float *rand, int size, int c, int h, int w, int crop_height, int crop_width, int train, int flip, float angle, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= size) return;

    float cx = w/2.;
    float cy = h/2.;

    int count = id;
    int j = id % crop_width;
    id /= crop_width;
    int i = id % crop_height;
    id /= crop_height;
    const int k = id % c;
    id /= c;
    int b = id;

    float r4 = rand[8*b + 4];
    float r5 = rand[8*b + 5];
    float r6 = rand[8*b + 6];
    float r7 = rand[8*b + 7];

    float dw = (w - crop_width)*r4;
    float dh = (h - crop_height)*r5;
    flip = (flip && (r6 > .5));
    angle = 2*angle*r7 - angle;
    if(!train){
        dw = (w - crop_width)/2.;
        dh = (h - crop_height)/2.;
        flip = 0;
        angle = 0;
    }

    input += w*h*c*b;

    float x = (flip) ? w - dw - j - 1 : j + dw;
    float y = i + dh;

    const float rx = cos(angle)*(x-cx) - sin(angle)*(y-cy) + cx;
    const float ry = sin(angle)*(x-cx) + cos(angle)*(y-cy) + cy;

    output[count] = bilinear_interpolate_kernel(input, w, h, rx, ry, k);
}

extern "C" void forward_crop_layer_gpu(crop_layer layer, network_state state)
{
    cuda_random(layer.rand_gpu, layer.batch*8);

    float radians = layer.angle*3.14159265/180.;

    float scale = 2;
    float translate = -1;
    if(layer.noadjust){
        scale = 1;
        translate = 0;
    }

    auto size = layer.batch * layer.w * layer.h;

    levels_image_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >>>(state.input, layer.rand_gpu, layer.batch, layer.w, layer.h, state.train, layer.saturation, layer.exposure, translate, scale, layer.shift);
    CHECK_CUDA(hipPeekAtLastError());

    size = layer.batch*layer.c*layer.out_w*layer.out_h;

    forward_crop_layer_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >>>(state.input, layer.rand_gpu, size, layer.c, layer.h, layer.w, layer.out_h, layer.out_w, state.train, layer.flip, radians, layer.output_gpu);
    CHECK_CUDA(hipPeekAtLastError());

/*
       cuda_pull_array(layer.output_gpu, layer.output, size);
       image im = float_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 0*(size/layer.batch));
       image im2 = float_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 1*(size/layer.batch));
       image im3 = float_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 2*(size/layer.batch));

       translate_image(im, -translate);
       scale_image(im, 1/scale);
       translate_image(im2, -translate);
       scale_image(im2, 1/scale);
       translate_image(im3, -translate);
       scale_image(im3, 1/scale);

       show_image(im, "cropped");
       show_image(im2, "cropped2");
       show_image(im3, "cropped3");
       cvWaitKey(0);
       */
}
